﻿/* Harjot Mangat
*  EECS 245 - Parallel Computing
*  Implementation of the kernel we covered in class
*/

#include "hip/hip_runtime.h"

//#include <device_functions.h>

#include <stdio.h>
#include <iostream>
//#include "kernel.h"

#define N 10000
#define BLOCK_SIZE 7
#define RADIUS 3
#define thread_count 55
#define block_count 10


float milliseconds = 0;



__global__ void stencil_1d(int *in, int *out) {
    __shared__ int temp[BLOCK_SIZE + 2*RADIUS];
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x + RADIUS;

    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS ) {
        temp[lindex - RADIUS] = in[gindex];
        temp[lindex + BLOCK_SIZE ] = in[gindex + BLOCK_SIZE];
    }

    __syncthreads();

    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++)
        result += temp[lindex + offset];

    out[gindex] = result;
}


void start_timer(hipEvent_t* start) {
    hipEventCreate(start);
    hipEventRecord(*start);
}


float stop_timer(hipEvent_t* start, hipEvent_t* stop) {
    hipEventCreate(stop);
    hipEventRecord(*stop);

    hipEventSynchronize(*stop);
    hipEventElapsedTime(&milliseconds, *start, *stop);
    return(milliseconds);
}


void cudaErrorCheck()
{
    hipError_t error = hipGetLastError();
    const char* errorName = hipGetErrorName(error);
    std::string s0(errorName);
    std::cout << ">>>> " << s0 << std::endl;

    const char* errorDescription = hipGetErrorString(error);
    std::string s1(errorDescription);
    std::cout << ">>>> " << s1 << std::endl;
}



void random_ints(int* a, int b)
{
    int i;
    for (i = 0; i < b; ++i)
        a[i] = rand() % 10;
}

//start of the main program
int main(void)
{
    int *in, *out;
    int *d_in, *d_out;
    int size = N * sizeof(int);
    hipEvent_t start, stop;

    //Allocating Memory for the device
    hipMalloc((void**)&d_in, size);
    hipMalloc((void**)&d_out, size);

    cudaErrorCheck();

    //Allocating Memory for the host and initialzing values for 'in' array
    in = (int*)malloc(size);
    random_ints(in, N);
    out = (int*)malloc(size);
    
    //copying host input memory to device memory
    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

    cudaErrorCheck();

    //starting the timer to check performance
    start_timer(&start);

    stencil_1d<<<N/block_count, thread_count>>>(d_in, d_out);


    std::cout << "Elapsed time: " << stop_timer(&start, &stop) << " ms" << std::endl;

    cudaErrorCheck();

    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

    cudaErrorCheck();

    free(in); free(out);
    hipFree(d_in); hipFree(d_out);

}