#include "hip/hip_runtime.h"
/* Harjot Mangat
*  EECS 245 - Parallel Computing
*  Implementation of the kernel we covered in class, implemented with texture memory
*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#endif

// Utilities and timing functions
#include "hip/hip_texture_types.h"
#include "texture_types.h"
#include <hip/hip_runtime.h> 
#include "hip/hip_runtime_api.h"


#define N 1000
#define BLOCK_SIZE 7
#define RADIUS 3
#define thread_count 5
#define block_count 1


float milliseconds = 0;

__global__ void stencil_1d(hipTextureObject_t textObj, int* out) {
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x + RADIUS;
    
    temp[lindex] = tex1Dfetch<int>(textObj,gindex);
    if (threadIdx.x < RADIUS) {
        temp[lindex - RADIUS] = tex1Dfetch<int>(textObj,gindex);
        temp[lindex + BLOCK_SIZE] = tex1Dfetch<int>(textObj, gindex + BLOCK_SIZE);
    }

    __syncthreads();

    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++)
        result += temp[lindex + offset];

    out[gindex] = result;
}


void start_timer(hipEvent_t* start) {
    hipEventCreate(start);
    hipEventRecord(*start);
}


float stop_timer(hipEvent_t* start, hipEvent_t* stop) {
    hipEventCreate(stop);
    hipEventRecord(*stop);

    hipEventSynchronize(*stop);
    hipEventElapsedTime(&milliseconds, *start, *stop);
    return(milliseconds);
}


void cudaErrorCheck()
{
    hipError_t error = hipGetLastError();
    const char* errorName = hipGetErrorName(error);
    std::string s0(errorName);
    std::cout << ">>>> " << s0 << std::endl;

    const char* errorDescription = hipGetErrorString(error);
    std::string s1(errorDescription);
    std::cout << ">>>> " << s1 << std::endl;
}



void random_ints(int* a, int b)
{
    int i;
    for (i = 0; i < b; ++i) {
        a[i] = rand() % 10;
    }
}

//start of the main program
int main(void)
{
    int* in, * out;
    
    int* d_in;
    int* d_out;
    int size = N * sizeof(int);
    hipEvent_t start, stop;

    //Allocating Memory for the device
    
    hipMalloc((void**)&d_in, size);
    hipMalloc((void**)&d_out, size);

    cudaErrorCheck();

    //Allocating Memory for the host and initialzing values for 'in' array
    in = (int*)malloc(size);
    random_ints(in, N);
    out = (int*)malloc(size);

    //copying host input memory to device memory
    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

    cudaErrorCheck();

    //building the ResourceDesc
    struct hipResourceDesc resDesc;
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = &d_in;
    resDesc.res.linear.sizeInBytes = N * sizeof(int);
    resDesc.res.linear.desc = hipCreateChannelDesc<int>();

    //building the TextureDesc
    struct hipTextureDesc texDesc = {};
    texDesc.readMode = hipReadModeElementType;

    //building the Texture
    hipTextureObject_t textObj;
    hipCreateTextureObject(&textObj, &resDesc, &texDesc, NULL);

    //starting the timer to check performance
    start_timer(&start);

    stencil_1d << <N/block_count, thread_count >> > (textObj, d_out);


    std::cout << "Elapsed time: " << stop_timer(&start, &stop) << " ms" << std::endl;

    cudaErrorCheck();

    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

    cudaErrorCheck();

    hipDestroyTextureObject(textObj);
    free(in); free(out);
    hipFree(d_in); hipFree(d_out);

}